#include "hip/hip_runtime.h"
#include <omp.h>
#include <spdlog/spdlog.h>

#include <algorithm>  // for std::min
#include <atomic>
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <mutex>
#include <thread>
#include <vector>

#include "app.hpp"
#include "cifar-dense/cuda/cu_dense_kernel.cuh"
#include "cifar-dense/dense_appdata.hpp"
#include "cifar-dense/omp/dense_kernel.hpp"
#include "common/cuda/cu_mem_resource.cuh"
#include "common/cuda/helpers.cuh"
#include "concurrentqueue.h"  // Moodycamel's ConcurrentQueue

// ---------------------------------------------------------------------
// Task structure
// ---------------------------------------------------------------------

struct Task {
  int uid;
  cifar_dense::AppData* appdata_ptr;
};

// Global atomic flag to control threads
std::atomic<bool> done(false);
// std::mutex mtx;

// ---------------------------------------------------------------------
// Producer
// ---------------------------------------------------------------------

void producer(moodycamel::ConcurrentQueue<Task>& queue,
              int num_tasks,
              std::vector<Task>& tasks,
              hipStream_t stream) {
  for (int i = 0; i < num_tasks; ++i) {
    // CUDA_CHECK(hipStreamAttachMemAsync(
    //     stream, tasks[i].u_data, 0, hipMemAttachHost));

    // u_image
    // u_conv1_weights
    // u_conv1_bias
    // u_conv1_out
    // u_pool1_out
    // u_conv2_weights
    // u_conv2_bias
    // u_conv2_out
    // u_pool2_out
    // u_conv3_weights
    // u_conv3_bias
    // u_conv3_out

    CUDA_CHECK(hipStreamAttachMemAsync(
        stream, tasks[i].appdata_ptr->u_image.data(), 0, hipMemAttachHost));
    CUDA_CHECK(
        hipStreamAttachMemAsync(stream,
                                 tasks[i].appdata_ptr->u_conv1_weights.data(),
                                 0,
                                 hipMemAttachHost));
    CUDA_CHECK(
        hipStreamAttachMemAsync(stream,
                                 tasks[i].appdata_ptr->u_conv1_bias.data(),
                                 0,
                                 hipMemAttachHost));
    CUDA_CHECK(
        hipStreamAttachMemAsync(stream,
                                 tasks[i].appdata_ptr->u_conv1_out.data(),
                                 0,
                                 hipMemAttachHost));
    CUDA_CHECK(
        hipStreamAttachMemAsync(stream,
                                 tasks[i].appdata_ptr->u_pool1_out.data(),
                                 0,
                                 hipMemAttachHost));
    CUDA_CHECK(
        hipStreamAttachMemAsync(stream,
                                 tasks[i].appdata_ptr->u_conv2_weights.data(),
                                 0,
                                 hipMemAttachHost));
    CUDA_CHECK(
        hipStreamAttachMemAsync(stream,
                                 tasks[i].appdata_ptr->u_conv2_bias.data(),
                                 0,
                                 hipMemAttachHost));
    CUDA_CHECK(
        hipStreamAttachMemAsync(stream,
                                 tasks[i].appdata_ptr->u_conv2_out.data(),
                                 0,
                                 hipMemAttachHost));
    CUDA_CHECK(
        hipStreamAttachMemAsync(stream,
                                 tasks[i].appdata_ptr->u_pool2_out.data(),
                                 0,
                                 hipMemAttachHost));
    CUDA_CHECK(
        hipStreamAttachMemAsync(stream,
                                 tasks[i].appdata_ptr->u_conv3_weights.data(),
                                 0,
                                 hipMemAttachHost));
    CUDA_CHECK(
        hipStreamAttachMemAsync(stream,
                                 tasks[i].appdata_ptr->u_conv3_bias.data(),
                                 0,
                                 hipMemAttachHost));
    CUDA_CHECK(
        hipStreamAttachMemAsync(stream,
                                 tasks[i].appdata_ptr->u_conv3_out.data(),
                                 0,
                                 hipMemAttachHost));

    CUDA_CHECK(hipStreamSynchronize(stream));

    // kernelA_CPU(&hostParams);

    auto g_little_core_size = g_little_cores.size();

#pragma omp parallel num_threads(g_little_core_size)
    {
      //       int thread_id = omp_get_thread_num();

      // #pragma omp critical
      //       printf("[OMP] Thread ID: %d\n", thread_id);

      cifar_dense::omp::process_stage_1(*tasks[i].appdata_ptr);
      cifar_dense::omp::process_stage_2(*tasks[i].appdata_ptr);
      cifar_dense::omp::process_stage_3(*tasks[i].appdata_ptr);
      cifar_dense::omp::process_stage_4(*tasks[i].appdata_ptr);
      cifar_dense::omp::process_stage_5(*tasks[i].appdata_ptr);
    }

    // Enqueue task
    queue.enqueue(tasks[i]);
  }

  // Signal consumer to stop
  done = true;
}

// ---------------------------------------------------------------------
// Consumer
// ---------------------------------------------------------------------

void consumer(moodycamel::ConcurrentQueue<Task>& queue, hipStream_t stream) {
  while (!done) {
    Task task;
    if (queue.try_dequeue(task)) {
      // CUDA_CHECK(hipStreamAttachMemAsync(
      //     stream, task.u_data, 0, hipMemAttachGlobal));

      CUDA_CHECK(hipStreamAttachMemAsync(stream,
                                          task.appdata_ptr->u_conv3_out.data(),
                                          0,
                                          hipMemAttachGlobal));
      CUDA_CHECK(
          hipStreamAttachMemAsync(stream,
                                   task.appdata_ptr->u_conv4_weights.data(),
                                   0,
                                   hipMemAttachGlobal));
      CUDA_CHECK(hipStreamAttachMemAsync(stream,
                                          task.appdata_ptr->u_conv4_bias.data(),
                                          0,
                                          hipMemAttachGlobal));
      CUDA_CHECK(hipStreamAttachMemAsync(stream,
                                          task.appdata_ptr->u_conv4_out.data(),
                                          0,
                                          hipMemAttachGlobal));

      CUDA_CHECK(
          hipStreamAttachMemAsync(stream,
                                   task.appdata_ptr->u_conv5_weights.data(),
                                   0,
                                   hipMemAttachGlobal));
      CUDA_CHECK(hipStreamAttachMemAsync(stream,
                                          task.appdata_ptr->u_conv5_bias.data(),
                                          0,
                                          hipMemAttachGlobal));
      CUDA_CHECK(hipStreamAttachMemAsync(stream,
                                          task.appdata_ptr->u_conv5_out.data(),
                                          0,
                                          hipMemAttachGlobal));

      CUDA_CHECK(hipStreamAttachMemAsync(stream,
                                          task.appdata_ptr->u_pool3_out.data(),
                                          0,
                                          hipMemAttachGlobal));
      CUDA_CHECK(
          hipStreamAttachMemAsync(stream,
                                   task.appdata_ptr->u_linear_weights.data(),
                                   0,
                                   hipMemAttachGlobal));

      CUDA_CHECK(
          hipStreamAttachMemAsync(stream,
                                   task.appdata_ptr->u_linear_bias.data(),
                                   0,
                                   hipMemAttachGlobal));
      CUDA_CHECK(hipStreamAttachMemAsync(stream,
                                          task.appdata_ptr->u_linear_out.data(),
                                          0,
                                          hipMemAttachGlobal));

      CUDA_CHECK(hipStreamSynchronize(stream));

      // // Process the task on GPU
      // kernelB_GPU<<<1, 256>>>(task.u_data, N);
      // kernelC_GPU<<<1, 256>>>(task.u_data, N);

      cifar_dense::cuda::process_stage_6(*task.appdata_ptr);
      cifar_dense::cuda::process_stage_7(*task.appdata_ptr);
      cifar_dense::cuda::process_stage_8(*task.appdata_ptr);
      cifar_dense::cuda::process_stage_9(*task.appdata_ptr);

    } else {
      // No task available, yield to avoid busy-waiting
      std::this_thread::yield();
    }
  }
}

// ---------------------------------------------------------------------
// 2 stage pipeline
// ---------------------------------------------------------------------

void run_2_stage() {
  moodycamel::ConcurrentQueue<Task> q_AB;
  const int num_tasks = 20;

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  // 1. prepare tasks
  auto mr = cuda::CudaMemoryResource();
  std::vector<Task> tasks(num_tasks);
  for (int i = 0; i < num_tasks; ++i) {
    tasks[i].uid = i;
    tasks[i].appdata_ptr = new cifar_dense::AppData(&mr);
  }

  // 2. Start producer and consumer threads

  auto start = std::chrono::high_resolution_clock::now();

  std::thread producer_thread(
      producer, std::ref(q_AB), num_tasks, std::ref(tasks), stream);
  std::thread consumer_thread(consumer, std::ref(q_AB), stream);

  // 3. Join threads
  producer_thread.join();
  consumer_thread.join();

  auto end = std::chrono::high_resolution_clock::now();
  auto total_ms =
      std::chrono::duration_cast<std::chrono::milliseconds>(end - start)
          .count();
  std::cout << " --- Total time taken: " << total_ms << " ms" << std::endl;
  std::cout << " --- Average time per task: " << total_ms / num_tasks << " ms"
            << std::endl;

  // 4. Free all pinned memory at the end
  for (int i = 0; i < num_tasks; ++i) {
    delete tasks[i].appdata_ptr;
  }

  CUDA_CHECK(hipStreamDestroy(stream));
}

// ---------------------------------------------------------------------
// Main
// ---------------------------------------------------------------------

int main(int argc, char* argv[]) {
  parse_args(argc, argv);

  run_2_stage();

  std::cout << "All tasks processed and memory freed." << std::endl;
  return 0;
}
