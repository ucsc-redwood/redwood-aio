#include <benchmark/benchmark.h>

#include "builtin-apps/app.hpp"
#include "builtin-apps/common/cuda/cu_mem_resource.cuh"
#include "builtin-apps/common/cuda/helpers.cuh"
#include "builtin-apps/tree/cuda/kernel.cuh"

#define PREPARE_DATA                    \
  auto mr = cuda::CudaMemoryResource(); \
  tree::AppData appdata(&mr);           \
  tree::cuda::TempStorage tmp_storage;  \
  CUDA_CHECK(hipDeviceSynchronize());

// ----------------------------------------------------------------
// Baseline
// ----------------------------------------------------------------

class CUDA_Tree : public benchmark::Fixture {};

BENCHMARK_DEFINE_F(CUDA_Tree, Baseline)
(benchmark::State& state) {
  PREPARE_DATA;

  for (auto _ : state) {
    tree::cuda::run_stage<1>(appdata, tmp_storage);
    tree::cuda::run_stage<2>(appdata, tmp_storage);
    tree::cuda::run_stage<3>(appdata, tmp_storage);
    tree::cuda::run_stage<4>(appdata, tmp_storage);
    tree::cuda::run_stage<5>(appdata, tmp_storage);
    tree::cuda::run_stage<6>(appdata, tmp_storage);
    tree::cuda::run_stage<7>(appdata, tmp_storage);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_Tree, Baseline)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 1
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_Tree, Stage1)
(benchmark::State& state) {
  PREPARE_DATA;

  // warmup
  tree::cuda::run_stage<1>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    tree::cuda::run_stage<1>(appdata, tmp_storage);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_Tree, Stage1)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 2
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_Tree, Stage2)
(benchmark::State& state) {
  PREPARE_DATA;

  tree::cuda::run_stage<1>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  // warmup
  tree::cuda::run_stage<2>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    tree::cuda::run_stage<2>(appdata, tmp_storage);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_Tree, Stage2)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 3
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_Tree, Stage3)
(benchmark::State& state) {
  PREPARE_DATA;

  tree::cuda::run_stage<1>(appdata, tmp_storage);
  tree::cuda::run_stage<2>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  // warmup
  tree::cuda::run_stage<3>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    tree::cuda::run_stage<3>(appdata, tmp_storage);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_Tree, Stage3)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 4
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_Tree, Stage4)
(benchmark::State& state) {
  PREPARE_DATA;

  tree::cuda::run_stage<1>(appdata, tmp_storage);
  tree::cuda::run_stage<2>(appdata, tmp_storage);
  tree::cuda::run_stage<3>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  // warmup
  tree::cuda::run_stage<4>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    tree::cuda::run_stage<4>(appdata, tmp_storage);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_Tree, Stage4)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 5
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_Tree, Stage5)
(benchmark::State& state) {
  PREPARE_DATA;

  tree::cuda::run_stage<1>(appdata, tmp_storage);
  tree::cuda::run_stage<2>(appdata, tmp_storage);
  tree::cuda::run_stage<3>(appdata, tmp_storage);
  tree::cuda::run_stage<4>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  // warmup
  tree::cuda::run_stage<5>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    tree::cuda::run_stage<5>(appdata, tmp_storage);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_Tree, Stage5)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 6
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_Tree, Stage6)
(benchmark::State& state) {
  PREPARE_DATA;

  tree::cuda::run_stage<1>(appdata, tmp_storage);
  tree::cuda::run_stage<2>(appdata, tmp_storage);
  tree::cuda::run_stage<3>(appdata, tmp_storage);
  tree::cuda::run_stage<4>(appdata, tmp_storage);
  tree::cuda::run_stage<5>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  // warmup
  tree::cuda::run_stage<6>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    tree::cuda::run_stage<6>(appdata, tmp_storage);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_Tree, Stage6)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 7
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_Tree, Stage7)
(benchmark::State& state) {
  PREPARE_DATA;

  tree::cuda::run_stage<1>(appdata, tmp_storage);
  tree::cuda::run_stage<2>(appdata, tmp_storage);
  tree::cuda::run_stage<3>(appdata, tmp_storage);
  tree::cuda::run_stage<4>(appdata, tmp_storage);
  tree::cuda::run_stage<5>(appdata, tmp_storage);
  tree::cuda::run_stage<6>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  // warmup
  tree::cuda::run_stage<7>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    tree::cuda::run_stage<7>(appdata, tmp_storage);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_Tree, Stage7)->Unit(benchmark::kMillisecond);

int main(int argc, char** argv) {
  spdlog::set_level(spdlog::level::off);

  benchmark::Initialize(&argc, argv);
  benchmark::RunSpecifiedBenchmarks();
  benchmark::Shutdown();
  return 0;
}