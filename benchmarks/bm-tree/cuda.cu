#include <benchmark/benchmark.h>

#include "../argc_argv_sanitizer.hpp"
#include "builtin-apps/app.hpp"
#include "builtin-apps/common/cuda/cu_mem_resource.cuh"
#include "builtin-apps/common/cuda/helpers.cuh"
#include "builtin-apps/resources_path.hpp"
#include "builtin-apps/tree/cuda/dispatchers.cuh"

#define PREPARE_DATA                    \
  auto mr = cuda::CudaMemoryResource(); \
  tree::AppData appdata(&mr);           \
  tree::cuda::TempStorage tmp_storage;  \
  CUDA_CHECK(hipDeviceSynchronize());

// ----------------------------------------------------------------
// Baseline
// ----------------------------------------------------------------

class CUDA_Tree : public benchmark::Fixture {};

BENCHMARK_DEFINE_F(CUDA_Tree, Baseline)
(benchmark::State& state) {
  PREPARE_DATA;

  for (auto _ : state) {
    tree::cuda::run_stage<1>(appdata, tmp_storage);
    tree::cuda::run_stage<2>(appdata, tmp_storage);
    tree::cuda::run_stage<3>(appdata, tmp_storage);
    tree::cuda::run_stage<4>(appdata, tmp_storage);
    tree::cuda::run_stage<5>(appdata, tmp_storage);
    tree::cuda::run_stage<6>(appdata, tmp_storage);
    tree::cuda::run_stage<7>(appdata, tmp_storage);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_Tree, Baseline)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 1
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_Tree, Stage1)
(benchmark::State& state) {
  PREPARE_DATA;

  // warmup
  tree::cuda::run_stage<1>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    tree::cuda::run_stage<1>(appdata, tmp_storage);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_Tree, Stage1)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 2
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_Tree, Stage2)
(benchmark::State& state) {
  PREPARE_DATA;

  tree::cuda::run_stage<1>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  // warmup
  tree::cuda::run_stage<2>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    tree::cuda::run_stage<2>(appdata, tmp_storage);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_Tree, Stage2)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 3
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_Tree, Stage3)
(benchmark::State& state) {
  PREPARE_DATA;

  tree::cuda::run_stage<1>(appdata, tmp_storage);
  tree::cuda::run_stage<2>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  // warmup
  tree::cuda::run_stage<3>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    tree::cuda::run_stage<3>(appdata, tmp_storage);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_Tree, Stage3)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 4
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_Tree, Stage4)
(benchmark::State& state) {
  PREPARE_DATA;

  tree::cuda::run_stage<1>(appdata, tmp_storage);
  tree::cuda::run_stage<2>(appdata, tmp_storage);
  tree::cuda::run_stage<3>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  // warmup
  tree::cuda::run_stage<4>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    tree::cuda::run_stage<4>(appdata, tmp_storage);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_Tree, Stage4)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 5
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_Tree, Stage5)
(benchmark::State& state) {
  PREPARE_DATA;

  tree::cuda::run_stage<1>(appdata, tmp_storage);
  tree::cuda::run_stage<2>(appdata, tmp_storage);
  tree::cuda::run_stage<3>(appdata, tmp_storage);
  tree::cuda::run_stage<4>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  // warmup
  tree::cuda::run_stage<5>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    tree::cuda::run_stage<5>(appdata, tmp_storage);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_Tree, Stage5)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 6
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_Tree, Stage6)
(benchmark::State& state) {
  PREPARE_DATA;

  tree::cuda::run_stage<1>(appdata, tmp_storage);
  tree::cuda::run_stage<2>(appdata, tmp_storage);
  tree::cuda::run_stage<3>(appdata, tmp_storage);
  tree::cuda::run_stage<4>(appdata, tmp_storage);
  tree::cuda::run_stage<5>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  // warmup
  tree::cuda::run_stage<6>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    tree::cuda::run_stage<6>(appdata, tmp_storage);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_Tree, Stage6)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 7
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_Tree, Stage7)
(benchmark::State& state) {
  PREPARE_DATA;

  tree::cuda::run_stage<1>(appdata, tmp_storage);
  tree::cuda::run_stage<2>(appdata, tmp_storage);
  tree::cuda::run_stage<3>(appdata, tmp_storage);
  tree::cuda::run_stage<4>(appdata, tmp_storage);
  tree::cuda::run_stage<5>(appdata, tmp_storage);
  tree::cuda::run_stage<6>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  // warmup
  tree::cuda::run_stage<7>(appdata, tmp_storage);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    tree::cuda::run_stage<7>(appdata, tmp_storage);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_Tree, Stage7)->Unit(benchmark::kMillisecond);

int main(int argc, char** argv) {
  parse_args(argc, argv);
  spdlog::set_level(spdlog::level::off);

  // Where to save the results json file?
  const auto storage_location = helpers::get_benchmark_storage_location();
  const auto out_name = storage_location.string() + "/BM_Tree_CUDA_" + g_device_id + ".json";

  // Sanitize the arguments to pass to Google Benchmark
  auto [new_argc, new_argv] = sanitize_argc_argv_for_benchmark(argc, argv, out_name);

  benchmark::Initialize(&new_argc, new_argv.data());
  if (benchmark::ReportUnrecognizedArguments(new_argc, new_argv.data())) return 1;
  benchmark::RunSpecifiedBenchmarks();
  benchmark::Shutdown();
  return 0;
}