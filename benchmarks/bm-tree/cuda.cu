#include <benchmark/benchmark.h>
#include <spdlog/spdlog.h>

#include <CLI/CLI.hpp>

#include "common/cuda/cu_mem_resource.cuh"
#include "common/cuda/helpers.cuh"
#include "tree/cuda/kernel.cuh"
#include "tree/tree_appdata.hpp"

// ----------------------------------------------------------------
// Baseline
// ----------------------------------------------------------------

class CUDA_Tree : public benchmark::Fixture {};

BENCHMARK_DEFINE_F(CUDA_Tree, Baseline)
(benchmark::State& state) {
  auto mr = cuda::CudaMemoryResource();
  tree::AppData app_data(&mr);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    tree::cuda::process_stage_1(app_data);
    tree::cuda::process_stage_2(app_data);
    tree::cuda::process_stage_3(app_data);
    tree::cuda::process_stage_4(app_data);
    tree::cuda::process_stage_5(app_data);
    tree::cuda::process_stage_6(app_data);
    tree::cuda::process_stage_7(app_data);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_Tree, Baseline)->Unit(benchmark::kMillisecond)->Iterations(10);

// ----------------------------------------------------------------
// Stage 1
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_Tree, Stage1)
(benchmark::State& state) {
  auto mr = cuda::CudaMemoryResource();
  tree::AppData app_data(&mr);

  // warmup
  tree::cuda::process_stage_1(app_data);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    tree::cuda::process_stage_1(app_data);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_Tree, Stage1)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 2
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_Tree, Stage2)
(benchmark::State& state) {
  auto mr = cuda::CudaMemoryResource();
  tree::AppData app_data(&mr);

  tree::cuda::process_stage_1(app_data);
  CUDA_CHECK(hipDeviceSynchronize());

  // warmup
  tree::cuda::process_stage_2(app_data);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    tree::cuda::process_stage_2(app_data);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_Tree, Stage2)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 3
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_Tree, Stage3)
(benchmark::State& state) {
  auto mr = cuda::CudaMemoryResource();
  tree::AppData app_data(&mr);

  tree::cuda::process_stage_1(app_data);
  tree::cuda::process_stage_2(app_data);
  CUDA_CHECK(hipDeviceSynchronize());

  // warmup
  tree::cuda::process_stage_3(app_data);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    tree::cuda::process_stage_3(app_data);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_Tree, Stage3)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 4
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_Tree, Stage4)
(benchmark::State& state) {
  auto mr = cuda::CudaMemoryResource();
  tree::AppData app_data(&mr);

  tree::cuda::process_stage_1(app_data);
  tree::cuda::process_stage_2(app_data);
  tree::cuda::process_stage_3(app_data);
  CUDA_CHECK(hipDeviceSynchronize());

  // warmup
  tree::cuda::process_stage_4(app_data);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    tree::cuda::process_stage_4(app_data);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_Tree, Stage4)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 5
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_Tree, Stage5)
(benchmark::State& state) {
  auto mr = cuda::CudaMemoryResource();
  tree::AppData app_data(&mr);

  tree::cuda::process_stage_1(app_data);
  tree::cuda::process_stage_2(app_data);
  tree::cuda::process_stage_3(app_data);
  tree::cuda::process_stage_4(app_data);
  CUDA_CHECK(hipDeviceSynchronize());

  // warmup
  tree::cuda::process_stage_5(app_data);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    tree::cuda::process_stage_5(app_data);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_Tree, Stage5)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 6
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_Tree, Stage6)
(benchmark::State& state) {
  auto mr = cuda::CudaMemoryResource();
  tree::AppData app_data(&mr);

  tree::cuda::process_stage_1(app_data);
  tree::cuda::process_stage_2(app_data);
  tree::cuda::process_stage_3(app_data);
  tree::cuda::process_stage_4(app_data);
  tree::cuda::process_stage_5(app_data);
  CUDA_CHECK(hipDeviceSynchronize());

  // warmup
  tree::cuda::process_stage_6(app_data);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    tree::cuda::process_stage_6(app_data);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_Tree, Stage6)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 7
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_Tree, Stage7)
(benchmark::State& state) {
  auto mr = cuda::CudaMemoryResource();
  tree::AppData app_data(&mr);

  tree::cuda::process_stage_1(app_data);
  tree::cuda::process_stage_2(app_data);
  tree::cuda::process_stage_3(app_data);
  tree::cuda::process_stage_4(app_data);
  tree::cuda::process_stage_5(app_data);
  tree::cuda::process_stage_6(app_data);
  CUDA_CHECK(hipDeviceSynchronize());

  // warmup
  tree::cuda::process_stage_7(app_data);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    tree::cuda::process_stage_7(app_data);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_Tree, Stage7)->Unit(benchmark::kMillisecond)->Iterations(10);

int main(int argc, char** argv) {
  spdlog::set_level(spdlog::level::off);

  benchmark::Initialize(&argc, argv);
  benchmark::RunSpecifiedBenchmarks();
  benchmark::Shutdown();

  tree::cuda::cleanup();

  return 0;
}