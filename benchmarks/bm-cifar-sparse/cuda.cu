#include <benchmark/benchmark.h>

#include "../argc_argv_sanitizer.hpp"
#include "builtin-apps/app.hpp"
#include "builtin-apps/cifar-sparse/cuda/cu_dispatcher.cuh"
#include "builtin-apps/cifar-sparse/sparse_appdata.hpp"
#include "builtin-apps/common/cuda/cu_mem_resource.cuh"
#include "builtin-apps/common/cuda/helpers.cuh"
#include "builtin-apps/resources_path.hpp"

#define PREPARE_DATA                    \
  auto mr = cuda::CudaMemoryResource(); \
  cifar_sparse::AppData appdata(&mr);   \
  CUDA_CHECK(hipDeviceSynchronize());

// ----------------------------------------------------------------
// Baseline
// ----------------------------------------------------------------

class CUDA_CifarSparse : public benchmark::Fixture {};

BENCHMARK_DEFINE_F(CUDA_CifarSparse, Baseline)
(benchmark::State& state) {
  PREPARE_DATA;

  for (auto _ : state) {
    cifar_sparse::cuda::run_stage<1>(appdata);
    cifar_sparse::cuda::run_stage<2>(appdata);
    cifar_sparse::cuda::run_stage<3>(appdata);
    cifar_sparse::cuda::run_stage<4>(appdata);
    cifar_sparse::cuda::run_stage<5>(appdata);
    cifar_sparse::cuda::run_stage<6>(appdata);
    cifar_sparse::cuda::run_stage<7>(appdata);
    cifar_sparse::cuda::run_stage<8>(appdata);
    cifar_sparse::cuda::run_stage<9>(appdata);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_CifarSparse, Baseline)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 1
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_CifarSparse, Stage1)
(benchmark::State& state) {
  PREPARE_DATA;

  // warmup
  cifar_sparse::cuda::run_stage<1>(appdata);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    cifar_sparse::cuda::run_stage<1>(appdata);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_CifarSparse, Stage1)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 2
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_CifarSparse, Stage2)
(benchmark::State& state) {
  PREPARE_DATA;

  cifar_sparse::cuda::run_stage<1>(appdata);
  // warmup
  cifar_sparse::cuda::run_stage<2>(appdata);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    cifar_sparse::cuda::run_stage<2>(appdata);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_CifarSparse, Stage2)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 3
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_CifarSparse, Stage3)
(benchmark::State& state) {
  PREPARE_DATA;

  cifar_sparse::cuda::run_stage<1>(appdata);
  cifar_sparse::cuda::run_stage<2>(appdata);
  CUDA_CHECK(hipDeviceSynchronize());

  // warmup
  cifar_sparse::cuda::run_stage<3>(appdata);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    cifar_sparse::cuda::run_stage<3>(appdata);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_CifarSparse, Stage3)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 4
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_CifarSparse, Stage4)
(benchmark::State& state) {
  PREPARE_DATA;

  cifar_sparse::cuda::run_stage<1>(appdata);
  cifar_sparse::cuda::run_stage<2>(appdata);
  cifar_sparse::cuda::run_stage<3>(appdata);
  CUDA_CHECK(hipDeviceSynchronize());

  cifar_sparse::cuda::run_stage<4>(appdata);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    cifar_sparse::cuda::run_stage<4>(appdata);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_CifarSparse, Stage4)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 5
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_CifarSparse, Stage5)
(benchmark::State& state) {
  PREPARE_DATA;

  cifar_sparse::cuda::run_stage<1>(appdata);
  cifar_sparse::cuda::run_stage<2>(appdata);
  cifar_sparse::cuda::run_stage<3>(appdata);
  cifar_sparse::cuda::run_stage<4>(appdata);
  CUDA_CHECK(hipDeviceSynchronize());

  // warmup
  cifar_sparse::cuda::run_stage<5>(appdata);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    cifar_sparse::cuda::run_stage<5>(appdata);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_CifarSparse, Stage5)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 6
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_CifarSparse, Stage6)
(benchmark::State& state) {
  PREPARE_DATA;

  cifar_sparse::cuda::run_stage<1>(appdata);
  cifar_sparse::cuda::run_stage<2>(appdata);
  cifar_sparse::cuda::run_stage<3>(appdata);
  cifar_sparse::cuda::run_stage<4>(appdata);
  cifar_sparse::cuda::run_stage<5>(appdata);
  CUDA_CHECK(hipDeviceSynchronize());

  // warmup
  cifar_sparse::cuda::run_stage<6>(appdata);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    cifar_sparse::cuda::run_stage<6>(appdata);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_CifarSparse, Stage6)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 7
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_CifarSparse, Stage7)
(benchmark::State& state) {
  PREPARE_DATA;

  cifar_sparse::cuda::run_stage<1>(appdata);
  cifar_sparse::cuda::run_stage<2>(appdata);
  cifar_sparse::cuda::run_stage<3>(appdata);
  cifar_sparse::cuda::run_stage<4>(appdata);
  cifar_sparse::cuda::run_stage<5>(appdata);
  cifar_sparse::cuda::run_stage<6>(appdata);
  CUDA_CHECK(hipDeviceSynchronize());

  // warmup
  cifar_sparse::cuda::run_stage<7>(appdata);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    cifar_sparse::cuda::run_stage<7>(appdata);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_CifarSparse, Stage7)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 8
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_CifarSparse, Stage8)
(benchmark::State& state) {
  PREPARE_DATA;

  cifar_sparse::cuda::run_stage<1>(appdata);
  cifar_sparse::cuda::run_stage<2>(appdata);
  cifar_sparse::cuda::run_stage<3>(appdata);
  cifar_sparse::cuda::run_stage<4>(appdata);
  cifar_sparse::cuda::run_stage<5>(appdata);
  cifar_sparse::cuda::run_stage<6>(appdata);
  cifar_sparse::cuda::run_stage<7>(appdata);
  CUDA_CHECK(hipDeviceSynchronize());

  // warmup
  cifar_sparse::cuda::run_stage<8>(appdata);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    cifar_sparse::cuda::run_stage<8>(appdata);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_CifarSparse, Stage8)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 9
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_CifarSparse, Stage9)
(benchmark::State& state) {
  PREPARE_DATA;

  cifar_sparse::cuda::run_stage<1>(appdata);
  cifar_sparse::cuda::run_stage<2>(appdata);
  cifar_sparse::cuda::run_stage<3>(appdata);
  cifar_sparse::cuda::run_stage<4>(appdata);
  cifar_sparse::cuda::run_stage<5>(appdata);
  cifar_sparse::cuda::run_stage<6>(appdata);
  cifar_sparse::cuda::run_stage<7>(appdata);
  cifar_sparse::cuda::run_stage<8>(appdata);
  CUDA_CHECK(hipDeviceSynchronize());

  // warmup
  cifar_sparse::cuda::run_stage<9>(appdata);
  CUDA_CHECK(hipDeviceSynchronize());

  for (auto _ : state) {
    cifar_sparse::cuda::run_stage<9>(appdata);
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

BENCHMARK_REGISTER_F(CUDA_CifarSparse, Stage9)->Unit(benchmark::kMillisecond);

int main(int argc, char** argv) {
  parse_args(argc, argv);
  spdlog::set_level(spdlog::level::off);

  // Where to save the results json file?
  const auto storage_location = helpers::get_benchmark_storage_location();
  const auto out_name = storage_location.string() + "/BM_CifarSparse_CUDA_" + g_device_id + ".json";

  // Sanitize the arguments to pass to Google Benchmark
  auto [new_argc, new_argv] = sanitize_argc_argv_for_benchmark(argc, argv, out_name);

  benchmark::Initialize(&new_argc, new_argv.data());
  if (benchmark::ReportUnrecognizedArguments(new_argc, new_argv.data())) return 1;
  benchmark::RunSpecifiedBenchmarks();
  benchmark::Shutdown();

  return 0;
}