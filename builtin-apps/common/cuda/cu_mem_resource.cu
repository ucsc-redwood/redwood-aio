#include <hip/hip_runtime.h>

#include "cu_mem_resource.cuh"
#include "helpers.cuh"

namespace cuda {

void* CudaMemoryResource::do_allocate(std::size_t bytes, std::size_t) {
  void* ptr = nullptr;
  CUDA_CHECK(hipMallocManaged(&ptr, bytes));
  return ptr;
}

void CudaMemoryResource::do_deallocate(void* p, std::size_t, std::size_t) {
  CUDA_CHECK(hipFree(p));
}

bool CudaMemoryResource::do_is_equal(const memory_resource& other) const noexcept {
  return dynamic_cast<const CudaMemoryResource*>(&other) != nullptr;
}

}  // namespace cuda
