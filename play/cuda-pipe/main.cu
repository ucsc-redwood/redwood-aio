#include "hip/hip_runtime.h"
#include <omp.h>

#include <algorithm>  // for std::min
#include <atomic>
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <mutex>
#include <thread>
#include <vector>

#include "concurrentqueue.h"  // Moodycamel's ConcurrentQueue

// Simple check macro
#define CHECK_CUDA(call)                                                   \
  do {                                                                     \
    hipError_t _status = call;                                            \
    if (_status != hipSuccess) {                                          \
      std::cerr << "Error: " << hipGetErrorString(_status) << " at line " \
                << __LINE__ << std::endl;                                  \
      exit(EXIT_FAILURE);                                                  \
    }                                                                      \
  } while (0)

// ---------------------------------------------------------------------
// GPU Kernels
// ---------------------------------------------------------------------

__global__ void kernelA_GPU(int* data, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    data[idx] = idx;
  }
}

__global__ void kernelB_GPU(int* data, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    data[idx] += 10;
  }
}

__global__ void kernelC_GPU(int* data, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    data[idx] *= 2;
  }
}

// ---------------------------------------------------------------------
// CPU (host) versions of the same "kernels."
// ---------------------------------------------------------------------
struct HostKernelParams {
  int* data;  // Points to pinned host memory
  int N;
};

static void kernelA_CPU(void* userData) {
  HostKernelParams* p = reinterpret_cast<HostKernelParams*>(userData);

#pragma omp parallel for
  for (int i = 0; i < p->N; i++) {
    p->data[i] = i;
  }
}

static void kernelB_CPU(void* userData) {
  HostKernelParams* p = reinterpret_cast<HostKernelParams*>(userData);

#pragma omp parallel for
  for (int i = 0; i < p->N; i++) {
    p->data[i] += 10;
  }
}

static void kernelC_CPU(void* userData) {
  HostKernelParams* p = reinterpret_cast<HostKernelParams*>(userData);

#pragma omp parallel for
  for (int i = 0; i < p->N; i++) {
    p->data[i] *= 2;
  }
}

// ---------------------------------------------------------------------
// Task structure
// ---------------------------------------------------------------------

struct Task {
  int uid;
  int* u_data;
};

// Global atomic flag to control threads
std::atomic<bool> done(false);
std::mutex mtx;

constexpr int N = 640 * 480;

// ---------------------------------------------------------------------
// Producer
// ---------------------------------------------------------------------

void producer(moodycamel::ConcurrentQueue<Task>& queue,
              int num_tasks,
              std::vector<Task>& tasks,
              hipStream_t stream) {
  // Produce tasks using the preallocated memory
  for (int i = 0; i < num_tasks; ++i) {
    // "Initialize" each preallocated task in host memory
    HostKernelParams hostParams;
    hostParams.data = tasks[i].u_data;
    hostParams.N = N;

    CHECK_CUDA(hipStreamAttachMemAsync(
        stream, tasks[i].u_data, 0, hipMemAttachHost));
    CHECK_CUDA(hipStreamSynchronize(stream));

    kernelA_CPU(&hostParams);

    // Enqueue task
    queue.enqueue(tasks[i]);
  }

  // Signal consumer to stop
  done = true;
}

// ---------------------------------------------------------------------
// Consumer
// ---------------------------------------------------------------------

void consumer(moodycamel::ConcurrentQueue<Task>& queue, hipStream_t stream) {
  while (!done) {
    Task task;
    if (queue.try_dequeue(task)) {
      CHECK_CUDA(hipStreamAttachMemAsync(
          stream, task.u_data, 0, hipMemAttachGlobal));
      CHECK_CUDA(hipStreamSynchronize(stream));

      // Process the task on GPU
      kernelB_GPU<<<1, 256>>>(task.u_data, N);
      kernelC_GPU<<<1, 256>>>(task.u_data, N);
    } else {
      // No task available, yield to avoid busy-waiting
      std::this_thread::yield();
    }
  }
}

// ---------------------------------------------------------------------
// Main
// ---------------------------------------------------------------------

int main(int argc, char* argv[]) {
  moodycamel::ConcurrentQueue<Task> queue;
  const int num_tasks = 20;

  hipStream_t stream;
  CHECK_CUDA(hipStreamCreate(&stream));

  // 1. Preallocate all tasks (along with pinned memory) at the beginning
  std::vector<Task> tasks(num_tasks);
  for (int i = 0; i < num_tasks; ++i) {
    int* u_data = nullptr;
    CHECK_CUDA(hipMallocManaged(&u_data, N * sizeof(int)));

    tasks[i].uid = i;
    tasks[i].u_data = u_data;
  }

  // 2. Start producer and consumer threads
  std::thread producer_thread(
      producer, std::ref(queue), num_tasks, std::ref(tasks), stream);
  std::thread consumer_thread(consumer, std::ref(queue), stream);

  // 3. Join threads
  producer_thread.join();
  consumer_thread.join();

  // 4. Free all pinned memory at the end
  for (int i = 0; i < num_tasks; ++i) {
    CHECK_CUDA(hipFree(tasks[i].u_data));
  }

  CHECK_CUDA(hipStreamDestroy(stream));

  std::cout << "All tasks processed and memory freed." << std::endl;
  return 0;
}
